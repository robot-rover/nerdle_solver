#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include "cuda_lib.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define NUM_SLOTS 8
#define SYMBOL_ORD 15
__device__ static constexpr char DEGUB_SYMBOL_TABLE[SYMBOL_ORD + 1] = "0123456789+-*/=";

__global__ void generate_clue_kernel(const uint8_t *secret, uint32_t num_secret, uint32_t secret_pitch, uint8_t *guess, uint32_t num_guess, uint32_t guess_pitch, uint8_t *clues, uint32_t clues_pitch)
{
    __shared__ uint8_t secret_cache[32 * NUM_SLOTS];
    __shared__ uint8_t guess_cache[32 * NUM_SLOTS];
    uint8_t counts[SYMBOL_ORD];
    uint8_t clue[NUM_SLOTS];

    uint32_t secret_addr = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t secret_offset = threadIdx.x * NUM_SLOTS;
    uint32_t guess_addr = threadIdx.y + blockDim.y * blockIdx.y;
    uint32_t guess_offset = threadIdx.y * NUM_SLOTS;
    int32_t clues_offset = secret_addr * NUM_SLOTS + guess_addr * clues_pitch;

    if (secret_addr < num_secret && guess_addr < num_guess)
    {
        if (threadIdx.y == 0)
        {
            uint32_t x_cache_offset = threadIdx.x * NUM_SLOTS;
            uint32_t x_main_offset = secret_addr * secret_pitch;
            for (int i = 0; i < NUM_SLOTS; i++)
            {
                secret_cache[x_cache_offset + i] = secret[x_main_offset + i];
            }
        }
        if (threadIdx.x == 0)
        {
            uint32_t y_cache_offset = threadIdx.y * NUM_SLOTS;
            uint32_t y_main_offset = guess_addr * guess_pitch;
            for (int i = 0; i < NUM_SLOTS; i++)
            {
                guess_cache[y_cache_offset + i] = guess[y_main_offset + i];
            }
        }
    }

    __syncthreads();

    if (secret_addr < num_secret && guess_addr < num_guess)
    {
        for (int i = 0; i < SYMBOL_ORD; i++)
        {
            counts[i] = 0;
        }
        for (int i = 0; i < NUM_SLOTS; i++)
        {
            // uint8_pitch = pitch / sizeof(uint8_t) => pitch

            uint8_t secret_symbol = secret_cache[secret_offset + i];
            uint8_t guess_symbol = guess_cache[guess_offset + i];
            uint8_t zero_if_green = secret_symbol == guess_symbol ? 0 : 1;
            counts[secret_symbol] += zero_if_green;
            clue[i] = zero_if_green;
        }
        for (int i = 0; i < NUM_SLOTS; i++)
        {
            uint8_t guess_symbol = guess_cache[secret_offset + i];
            uint8_t one_if_yellow = clue[i] & (counts[guess_symbol] > 0 ? 1 : 0);
            counts[guess_symbol] -= one_if_yellow;
            clue[i] += one_if_yellow;
        }
        for (int i = 0; i < NUM_SLOTS; i++)
        {
            clues[clues_offset + i] = clue[i];
        }
    }
}

extern "C" ClueContext* create_context(uint32_t num_secret, uint32_t num_guess) {
    ClueContext *ctx = (ClueContext*)malloc(sizeof(ClueContext));
    ctx->secret_alloc_rows = num_secret;
    ctx->guess_alloc_rows = num_guess;

    // Size of host eqs (and retval clues)
    size_t eq_width = sizeof(uint8_t) * NUM_SLOTS;

    // Create guess on device (2D)
    gpuErrchk(hipMallocPitch((void **)&ctx->d_guess, &ctx->guess_pitch, eq_width, num_guess));

    // Create secret on device (2D)
    gpuErrchk(hipMallocPitch((void **)&ctx->d_secret, &ctx->secret_pitch, eq_width, num_secret));

    // Create clues on device (2D)
    gpuErrchk(hipMallocPitch((void **)&ctx->d_clues, &ctx->clues_pitch, eq_width * num_secret, num_guess));

    return ctx;
}

extern "C" void free_context(ClueContext *ctx) {
    // Free device global memory
    gpuErrchk(hipFree(ctx->d_secret));
    gpuErrchk(hipFree(ctx->d_guess));
    gpuErrchk(hipFree(ctx->d_clues));
    free(ctx);
}

extern "C" int generate_clueg(ClueContext *ctx, uint8_t *secret_eqs, uint32_t num_secret, uint8_t *guess_eqs, uint32_t num_guess, uint8_t *clue_arr)
{
    size_t eq_width = sizeof(uint8_t) * NUM_SLOTS;
    if (num_guess > ctx->guess_alloc_rows) {
        return -1;
    }
    if (num_secret > ctx->secret_alloc_rows) {
        return -2;
    }

    // Copy to the device
    gpuErrchk(hipMemcpy2D(ctx->d_secret, ctx->secret_pitch, secret_eqs, eq_width, eq_width, num_secret, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy2D(ctx->d_guess, ctx->guess_pitch, guess_eqs, eq_width, eq_width, num_guess, hipMemcpyHostToDevice));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 32;
    int blocksPerGridx = (num_secret + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGridy = (num_guess + threadsPerBlock - 1) / threadsPerBlock;
    dim3 blocks(blocksPerGridx, blocksPerGridy);
    dim3 threads(threadsPerBlock, threadsPerBlock);

    // printf("CUDA: kernel launch with %dx%d blocks of %d threads\n", blocksPerGridx, blocksPerGridy, threadsPerBlock);
    generate_clue_kernel<<<blocks, threads>>>(ctx->d_secret, num_secret, ctx->secret_pitch, ctx->d_guess, num_guess, ctx->guess_pitch, ctx->d_clues, ctx->clues_pitch);

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    gpuErrchk(hipMemcpy2D(clue_arr, eq_width * num_secret, ctx->d_clues, ctx->clues_pitch, eq_width * num_secret, num_guess, hipMemcpyDeviceToHost));

    return 0;
}

extern "C" void helloworld() {
    printf("Hello World\n");
}